#include "hip/hip_runtime.h"
#include <stdio.h>   // printf
#include <stdlib.h>  // EXIT_FAILURE

#include "cupdlp_cudalinalg.cuh"

inline int nBlocks256(int n) {
  constexpr int BLOCKS_PER_SM = 32;
  int numSMs;
  CHECK_CUDA_IGNORE(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0))
  return min((n + 256 - 1) / 256, BLOCKS_PER_SM * numSMs);
}

extern "C" {

cupdlp_int cuda_alloc_MVbuffer(
    hipsparseHandle_t handle, hipsparseSpMatDescr_t cuda_csc,
    hipsparseDnVecDescr_t vecX, hipsparseDnVecDescr_t vecAx,
    hipsparseSpMatDescr_t cuda_csr, hipsparseDnVecDescr_t vecY,
    hipsparseDnVecDescr_t vecATy, void **dBuffer_csc_ATy, void **dBuffer_csr_Ax) {

  size_t AxBufferSize = 0;
  size_t ATyBufferSize = 0;
  cupdlp_float alpha = 1.0;
  cupdlp_float beta = 0.0;
  // hipsparseSpSVAlg_t alg = HIPSPARSE_SPSV_ALG_DEFAULT;
  hipsparseSpMVAlg_t alg = HIPSPARSE_SPMV_CSR_ALG2; //deterministic

  // get the buffer size needed by csr Ax
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, cuda_csr, vecX, &beta,
      vecAx, CudaComputeType, alg, &AxBufferSize))

  // allocate an external buffer if needed
  CHECK_CUDA(hipMalloc(dBuffer_csr_Ax, AxBufferSize))

  // get the buffer size needed by csc ATy
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
      handle, HIPSPARSE_OPERATION_TRANSPOSE, &alpha, cuda_csc, vecY, &beta,
      vecATy, CudaComputeType, alg, &ATyBufferSize))

  // allocate an external buffer if needed
  CHECK_CUDA(hipMalloc(dBuffer_csc_ATy, ATyBufferSize))

  return EXIT_SUCCESS;
}

/*
cupdlp_int cuda_csc_Ax(hipsparseHandle_t handle,
                       hipsparseSpMatDescr_t cuda_csc,
                       hipsparseDnVecDescr_t vecX,
                       hipsparseDnVecDescr_t vecAx, void *dBuffer,
                       cupdlp_float alpha, cupdlp_float beta) {
  // Ax = alpha * Acsc * X + beta * Ax

  hipsparseOperation_t op = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  CHECK_CUSPARSE(hipsparseSpMV(handle, op, &alpha, cuda_csc, vecX, &beta, vecAx,
                              // CudaComputeType, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                              CudaComputeType, HIPSPARSE_SPMV_CSR_ALG2, dBuffer))

  return EXIT_SUCCESS;
}
*/

cupdlp_int cuda_csr_Ax(hipsparseHandle_t handle,
                       hipsparseSpMatDescr_t cuda_csr,
                       hipsparseDnVecDescr_t vecX,
                       hipsparseDnVecDescr_t vecAx, void *dBuffer,
                       cupdlp_float alpha, cupdlp_float beta) {
  // Ax = alpha * Acsr * X + beta * Ax

  hipsparseOperation_t op = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  CHECK_CUSPARSE(hipsparseSpMV(handle, op, &alpha, cuda_csr, vecX, &beta, vecAx,
                              // CudaComputeType, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                              CudaComputeType, HIPSPARSE_SPMV_CSR_ALG2, dBuffer))

  return EXIT_SUCCESS;
}

cupdlp_int cuda_csc_ATy(hipsparseHandle_t handle,
                        hipsparseSpMatDescr_t cuda_csc,
                        hipsparseDnVecDescr_t vecY,
                        hipsparseDnVecDescr_t vecATy, void *dBuffer,
                        cupdlp_float alpha, cupdlp_float beta) {
  // ATy = alpha * Acsc^T * Y + beta * ATy
  hipsparseOperation_t op = HIPSPARSE_OPERATION_TRANSPOSE;

  CHECK_CUSPARSE(hipsparseSpMV(handle, op, &alpha, cuda_csc, vecY, &beta, vecATy,
                              // CudaComputeType, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                              CudaComputeType, HIPSPARSE_SPMV_CSR_ALG2, dBuffer))

  return EXIT_SUCCESS;
}

/*
cupdlp_int cuda_csr_ATy(hipsparseHandle_t handle,
                        hipsparseSpMatDescr_t cuda_csr,
                        hipsparseDnVecDescr_t vecY,
                        hipsparseDnVecDescr_t vecATy, void *dBuffer,
                        cupdlp_float alpha, cupdlp_float beta) {
  // ATy = alpha * Acsr^T * Y + beta * ATy
  hipsparseOperation_t op = HIPSPARSE_OPERATION_TRANSPOSE;

  CHECK_CUSPARSE(hipsparseSpMV(handle, op, &alpha, cuda_csr, vecY, &beta, vecATy,
                              // CudaComputeType, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                              CudaComputeType, HIPSPARSE_SPMV_CSR_ALG2, dBuffer))

  return EXIT_SUCCESS;
}
*/

void cupdlp_primal_feasibility_kernel_cuda(cupdlp_float *z,
                                           const cupdlp_float *ax,
                                           const cupdlp_float *rhs,
                                           const cupdlp_float *rowScale,
                                           int ifScaled,
                                           int nEqs, int nRows) {
  element_primal_feas_kernel<<<nBlocks256(nRows), 256>>>(z, ax, rhs, rowScale,
                                                         ifScaled, nEqs, nRows);
}

void cupdlp_dual_feasibility_kernel_1_cuda(cupdlp_float *z,
                                           const cupdlp_float *aty,
                                           const cupdlp_float *cost,
                                           int nCols) {
  element_dual_feas_kernel_1<<<nBlocks256(nCols), 256>>>(z, aty, cost, nCols);
}

void cupdlp_dual_feasibility_kernel_2_cuda(cupdlp_float *z,
                                           const cupdlp_float *dualResidual,
                                           const cupdlp_float *hasLower,
                                           int nCols) {
  element_dual_feas_kernel_2<<<nBlocks256(nCols), 256>>>(z, dualResidual,
                                                         hasLower, nCols);
}

void cupdlp_dual_feasibility_kernel_3_cuda(cupdlp_float *z,
                                           const cupdlp_float *dualResidual,
                                           const cupdlp_float *hasUpper,
                                           int nCols) {
  element_dual_feas_kernel_3<<<nBlocks256(nCols), 256>>>(z, dualResidual,
                                                         hasUpper, nCols);
}

void cupdlp_primal_infeasibility_kernel_cuda(cupdlp_float *z, const cupdlp_float *aty,
                                             const cupdlp_float *dSlackPos,
                                             const cupdlp_float *dSlackNeg,
                                             const cupdlp_float *colScale,
                                             cupdlp_float alpha, int ifScaled, int nCols) {
  element_primal_infeas_kernel<<<nBlocks256(nCols), 256>>>(z, aty, dSlackPos,
                                                           dSlackNeg, colScale, alpha,
                                                           ifScaled, nCols);
}

void cupdlp_dual_infeasibility_kernel_lb_cuda(cupdlp_float *z, const cupdlp_float *x,
                                              const cupdlp_float *hasLower,
                                              const cupdlp_float *colScale,
                                              cupdlp_float alpha, int ifScaled, int nCols) {
  element_dual_infeas_kernel_lb<<<nBlocks256(nCols), 256>>>(z, x, hasLower,
                                                            colScale, alpha,
                                                            ifScaled, nCols);
}

void cupdlp_dual_infeasibility_kernel_ub_cuda(cupdlp_float *z, const cupdlp_float *x,
                                              const cupdlp_float *hasUpper,
                                              const cupdlp_float *colScale,
                                              cupdlp_float alpha, int ifScaled, int nCols) {
  element_dual_infeas_kernel_ub<<<nBlocks256(nCols), 256>>>(z, x, hasUpper,
                                                            colScale, alpha,
                                                            ifScaled, nCols);
}

void cupdlp_dual_infeasibility_kernel_constr_cuda(cupdlp_float *z, const cupdlp_float *ax,
                                                  const cupdlp_float *rowScale,
                                                  cupdlp_float alpha, int ifScaled,
                                                  int nEqs, int nRows) {
  element_dual_infeas_kernel_constr<<<nBlocks256(nRows), 256>>>(z, ax, rowScale,
                                                                alpha, ifScaled, nEqs, nRows);
}

void cupdlp_projSameub_cuda(cupdlp_float *x, cupdlp_float ub, int n) {
  element_wise_projSameub_kernel<<<nBlocks256(n), 256>>>(x, ub, n);
}

void cupdlp_projSamelb_cuda(cupdlp_float *x, cupdlp_float lb, int n) {
  element_wise_projSamelb_kernel<<<nBlocks256(n), 256>>>(x, lb, n);
}

void cupdlp_projub_cuda(cupdlp_float *x, const cupdlp_float *ub, int n) {
  element_wise_projub_kernel<<<nBlocks256(n), 256>>>(x, ub, n);
}

void cupdlp_projlb_cuda(cupdlp_float *x, const cupdlp_float *lb, int n) {
  element_wise_projlb_kernel<<<nBlocks256(n), 256>>>(x, lb, n);
}

void cupdlp_ediv_cuda(cupdlp_float *x, const cupdlp_float *y, int n) {
  element_wise_div_kernel<<<nBlocks256(n), 256>>>(x, y, n);
}

void cupdlp_edot_cuda(cupdlp_float *x, const cupdlp_float *y, int n) {
  element_wise_dot_kernel<<<nBlocks256(n), 256>>>(x, y, n);
}

void cupdlp_haslb_cuda(cupdlp_float *haslb, const cupdlp_float *lb,
                       cupdlp_float bound, int n) {
  element_wise_initHaslb_kernel<<<nBlocks256(n), 256>>>(haslb, lb, bound, n);
}

void cupdlp_hasub_cuda(cupdlp_float *hasub, const cupdlp_float *ub,
                       cupdlp_float bound, int n) {
  element_wise_initHasub_kernel<<<nBlocks256(n), 256>>>(hasub, ub, bound, n);
}

void cupdlp_filterlb_cuda(cupdlp_float *x, const cupdlp_float *lb,
                          cupdlp_float bound, int n) {
  element_wise_filterlb_kernel<<<nBlocks256(n), 256>>>(x, lb, bound, n);
}

void cupdlp_filterub_cuda(cupdlp_float *x, const cupdlp_float *ub,
                          cupdlp_float bound, int n) {
  element_wise_filterub_kernel<<<nBlocks256(n), 256>>>(x, ub, bound, n);
}

void cupdlp_initvec_cuda(cupdlp_float *x, cupdlp_float val, int n) {
  init_cuda_vec_kernel<<<nBlocks256(n), 256>>>(x, val, n);
}

void cupdlp_pgrad_cuda(cupdlp_float *xUpdate, const cupdlp_float *x,
                       const cupdlp_float *cost, const cupdlp_float *ATy,
                       const cupdlp_float *lb, const cupdlp_float *ub,
                       cupdlp_float dPrimalStep, int nCols) {
  primal_grad_step_kernel<<<nBlocks256(nCols), 256>>>(xUpdate, x, cost, ATy, lb, ub, dPrimalStep, nCols);
}

void cupdlp_dgrad_cuda(cupdlp_float *yUpdate,
                       const cupdlp_float *y, const cupdlp_float *b,
                       const cupdlp_float *Ax, const cupdlp_float *AxUpdate,
                       cupdlp_float dDualStep, int nRows, int nEqs) {
  dual_grad_step_kernel<<<nBlocks256(nRows), 256>>>(yUpdate, y, b, Ax, AxUpdate, dDualStep, nRows, nEqs);
}

/*
void cupdlp_sub_cuda(cupdlp_float *z, const cupdlp_float *x,
                     const cupdlp_float *y, int n)
{
  naive_sub_kernel<<<nBlocks256(n), 256>>>(z, x, y, n);
}
*/


void cupdlp_movement_interaction_cuda(
    cupdlp_float *dX2, cupdlp_float *dY2, cupdlp_float *dInter, cupdlp_float *buffer,
    const cupdlp_float *xUpdate, const cupdlp_float *x,
    const cupdlp_float *yUpdate, const cupdlp_float *y,
    const cupdlp_float *atyUpdate, const cupdlp_float *aty,
    int nRows, int nCols)
{
  int warpSize;
  CHECK_CUDA_IGNORE(hipDeviceGetAttribute(&warpSize, hipDeviceAttributeWarpSize, 0))
  if (warpSize != 32) {
    printf("warpSize\n");
    exit(1);
  }

  constexpr int RED_BLOCK_SIZE = 256;
  constexpr int RED_ELS_PER_THREAD = 4;
  constexpr int RED_ELS_PER_BLOCK = RED_BLOCK_SIZE * RED_ELS_PER_THREAD;

  constexpr int SUM_BLOCK_SIZE = 512;
  constexpr int SUM_ELS_PER_THREAD = 2;
  constexpr int SUM_ELS_PER_BLOCK = SUM_BLOCK_SIZE * SUM_ELS_PER_THREAD;

  int nBlocksCols = (nCols + RED_ELS_PER_BLOCK - 1) / RED_ELS_PER_BLOCK;
  int nBlocksRows = (nRows + RED_ELS_PER_BLOCK - 1) / RED_ELS_PER_BLOCK;

  int buf_size = 256 * ((max(nBlocksCols,nBlocksRows) + 256 - 1) / 256);

  // buffer has size max(nCols, nRows, 2048)

  // we have buf_size <= max(nBlocksCols, nBlocksRows) + 256 - 1
  //                  <= max(nCols, nRows) / (4 * 256) + 1 + 256 - 1,
  // since RED_ELS_PER_BLOCK = 4 * 256

  // assume wlog that nRows <= nCols

  // if nCols <= 2 * 4 * 256, then
  // buf_size <= 2 + 256 <= 258,
  // so 5 * buf_size <= 1290 <= max(nCols, nRows, 2048)

  // if nCols > 2 * 4 * 256, then
  // buf_size <= nCols / (4 * 256) + nCols / 8 <= nCols * 3/16
  // so 5 * buf_size <= 15/16 * nCols <= max(nCols, nRows, 2048)

  cupdlp_float *buf_1 = buffer + 0 * buf_size;
  cupdlp_float *buf_2 = buffer + 1 * buf_size;
  cupdlp_float *buf_3 = buffer + 2 * buf_size;
  cupdlp_float *buf_4 = buffer + 3 * buf_size;
  cupdlp_float *buf_5 = buffer + 4 * buf_size;

  int nBlocks = nBlocksCols;
  movement_1_kernel<<<nBlocks, RED_BLOCK_SIZE>>>(buf_1, buf_2, xUpdate, x, atyUpdate, aty, nCols);

  while (nBlocks > 1) {
    int nBlocks2 = (nBlocks + SUM_ELS_PER_BLOCK - 1) / SUM_ELS_PER_BLOCK;
    sum_kernel<<<nBlocks2, SUM_BLOCK_SIZE>>>(buf_3, buf_1, nBlocks);
    sum_kernel<<<nBlocks2, SUM_BLOCK_SIZE>>>(buf_4, buf_2, nBlocks);
    nBlocks = nBlocks2;
    cupdlp_float *tmp = buf_1;
    buf_1 = buf_3;
    buf_3 = tmp;
    tmp = buf_2;
    buf_2 = buf_4;
    buf_4 = tmp;
  }

  CHECK_CUDA_STRICT(hipMemcpyAsync(buf_5 + 0, buf_1, sizeof(cupdlp_float), hipMemcpyDeviceToDevice))
  CHECK_CUDA_STRICT(hipMemcpyAsync(buf_5 + 1, buf_2, sizeof(cupdlp_float), hipMemcpyDeviceToDevice))

  nBlocks = nBlocksRows;
  movement_2_kernel<<<nBlocks, RED_BLOCK_SIZE>>>(buf_1, yUpdate, y, nRows);

  while (nBlocks > 1) {
    int nBlocks2 = (nBlocks + SUM_ELS_PER_BLOCK - 1) / SUM_ELS_PER_BLOCK;
    sum_kernel<<<nBlocks2, SUM_BLOCK_SIZE>>>(buf_2, buf_1, nBlocks);
    nBlocks = nBlocks2;
    cupdlp_float *tmp = buf_1;
    buf_1 = buf_2;
    buf_2 = tmp;
  }

  cupdlp_float res[3];
  CHECK_CUDA_STRICT(hipMemcpyAsync(buf_5 + 2, buf_1, sizeof(cupdlp_float), hipMemcpyDeviceToDevice))
  CHECK_CUDA_STRICT(hipDeviceSynchronize())
  CHECK_CUDA_STRICT(hipMemcpy(res, buf_5, 3 * sizeof(cupdlp_float), hipMemcpyDeviceToHost))
  CHECK_CUDA_LAST();

  *dX2 = res[0];
  *dY2 = res[2];
  *dInter = res[1];
}

cupdlp_int print_cuda_info(hipsparseHandle_t handle)
{
#if PRINT_CUDA_INFO

  int v_cuda_runtime = 0;
  int v_cuda_driver = 0;
  int v_cusparse = 0;
  CHECK_CUDA(hipRuntimeGetVersion(&v_cuda_runtime))
  CHECK_CUDA(hipDriverGetVersion(&v_cuda_driver))
  CHECK_CUSPARSE(hipsparseGetVersion(handle, &v_cusparse))

  printf("Cuda runtime %d\n", v_cuda_runtime);
  printf("Cuda driver %d\n", v_cuda_driver);
  printf("cuSparse %d\n", v_cusparse);

  int n_devices = 0;
  CHECK_CUDA(hipGetDeviceCount(&n_devices))

  for (int i = 0; i < n_devices; i++) {
    hipDeviceProp_t prop;
    CHECK_CUDA(hipGetDeviceProperties(&prop, i));

    printf("Cuda device %d: %s\n", i, prop.name);
#if PRINT_DETAILED_CUDA_INFO
    printf("  Clock rate (KHz): %d\n", prop.clockRate);
    printf("  Memory clock rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory bus width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak memory bandwidth (GB/s): %f\n",
            2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    printf("  Global memory available on device (GB): %f\n", prop.totalGlobalMem / 1.0e9);
    printf("  Shared memory available per block (B): %zu\n", prop.sharedMemPerBlock);
    printf("  Warp size in threads: %d\n", prop.warpSize);
    printf("  Maximum number of threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("  Number of multiprocessors on device: %d\n", prop.multiProcessorCount);
#endif
  }
#endif

  return EXIT_SUCCESS;
}

}
