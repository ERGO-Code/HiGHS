#include "hip/hip_runtime.h"
#include "cupdlp_cuda_kernels.cuh"



__global__ void element_primal_feas_kernel(cupdlp_float *z,
                                           const cupdlp_float *ax,
                                           const cupdlp_float *rhs,
                                           const cupdlp_float *rowScale,
                                           int ifScaled,
                                           int nEqs, int nRows) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nRows; i += gridDim.x * blockDim.x) {
    cupdlp_float tmp = ax[i] - rhs[i];
    if (i >= nEqs) tmp = min(tmp, 0.0);
    z[i] = tmp * (ifScaled ? rowScale[i] : 1.0);
  }
}

__global__ void element_dual_feas_kernel_1(cupdlp_float *z,
                                           const cupdlp_float *aty,
                                           const cupdlp_float *cost,
                                           int nCols) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nCols; i += gridDim.x * blockDim.x) {
    z[i] = cost[i] - aty[i];
  }
}

__global__ void element_dual_feas_kernel_2(cupdlp_float *z,
                                           const cupdlp_float *dualResidual,
                                           const cupdlp_float *hasLower,
                                           int nCols) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nCols; i += gridDim.x * blockDim.x) {
    z[i] = max(dualResidual[i], 0.0) * hasLower[i];
  }
}

__global__ void element_dual_feas_kernel_3(cupdlp_float *z,
                                           const cupdlp_float *dualResidual,
                                           const cupdlp_float *hasUpper,
                                           int nCols) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nCols; i += gridDim.x * blockDim.x) {
    z[i] = -min(dualResidual[i], 0.0) * hasUpper[i];
  }
}

__global__ void element_primal_infeas_kernel(cupdlp_float *z, const cupdlp_float *aty,
                                             const cupdlp_float *dSlackPos,
                                             const cupdlp_float *dSlackNeg,
                                             const cupdlp_float *colScale,
                                             cupdlp_float alpha, int ifScaled, int nCols) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nCols; i += gridDim.x * blockDim.x) {
    z[i] = alpha * (aty[i] + dSlackPos[i] - dSlackNeg[i]) * (ifScaled ? colScale[i] : 1.0);
  }
}

__global__ void element_dual_infeas_kernel_lb(cupdlp_float *z,
                                              const cupdlp_float *x,
                                              const cupdlp_float *hasLower,
                                              const cupdlp_float *colScale,
                                              cupdlp_float alpha, int ifScaled, int nCols) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nCols; i += gridDim.x * blockDim.x) {
    z[i] = min(alpha * x[i], 0.0) * hasLower[i] / (ifScaled ? colScale[i] : 1.0);
  }
}

__global__ void element_dual_infeas_kernel_ub(cupdlp_float *z,
                                              const cupdlp_float *x,
                                              const cupdlp_float *hasUpper,
                                              const cupdlp_float *colScale,
                                              cupdlp_float alpha, int ifScaled, int nCols) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nCols; i += gridDim.x * blockDim.x) {
    z[i] = max(alpha * x[i], 0.0) * hasUpper[i] / (ifScaled ? colScale[i] : 1.0);
  }
}

__global__ void element_dual_infeas_kernel_constr(cupdlp_float *z,
                                                  const cupdlp_float *ax,
                                                  const cupdlp_float *rowScale,
                                                  cupdlp_float alpha, int ifScaled,
                                                  int nEqs, int nRows) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nRows; i += gridDim.x * blockDim.x) {
    cupdlp_float tmp = alpha * ax[i];
    if (i >= nEqs) tmp = min(tmp, 0.0);
    z[i] = tmp * (ifScaled ? rowScale[i] : 1.0);
  }
}

__global__ void element_wise_dot_kernel(cupdlp_float *x, const cupdlp_float *y, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    x[i] *= y[i];
  }
}

__global__ void element_wise_div_kernel(cupdlp_float *x, const cupdlp_float *y, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    x[i] /= y[i];
  }
}

__global__ void element_wise_projlb_kernel(cupdlp_float *x,
                                           const cupdlp_float *lb, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    x[i] = max(x[i], lb[i]);
  }
}

__global__ void element_wise_projub_kernel(cupdlp_float *x,
                                           const cupdlp_float *ub, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    x[i] = min(x[i], ub[i]);
  }
}

__global__ void element_wise_projSamelb_kernel(cupdlp_float *x,
                                               cupdlp_float lb, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    x[i] = max(x[i], lb);
  }
}

__global__ void element_wise_projSameub_kernel(cupdlp_float *x,
                                               cupdlp_float ub, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    x[i] = min(x[i], ub);
  }
}

__global__ void element_wise_initHaslb_kernel(cupdlp_float *haslb,
                                              const cupdlp_float *lb,
                                              cupdlp_float bound, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    haslb[i] = lb[i] > bound ? 1.0 : 0.0;
  }
}

__global__ void element_wise_initHasub_kernel(cupdlp_float *hasub,
                                              const cupdlp_float *ub,
                                              cupdlp_float bound, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    hasub[i] = ub[i] < bound ? 1.0 : 0.0;
  }
}

__global__ void element_wise_filterlb_kernel(cupdlp_float *x,
                                             const cupdlp_float *lb,
                                             cupdlp_float bound, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    x[i] = lb[i] > bound ? lb[i] : 0.0;
  }
}

__global__ void element_wise_filterub_kernel(cupdlp_float *x,
                                             const cupdlp_float *ub,
                                             cupdlp_float bound, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    x[i] = ub[i] < bound ? ub[i] : 0.0;
  }
}

__global__ void init_cuda_vec_kernel(cupdlp_float *x, cupdlp_float val, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    x[i] = val;
  }
}

// xUpdate = proj(x - dPrimalStep * (cost - ATy))
__global__ void primal_grad_step_kernel(cupdlp_float *__restrict__ xUpdate,
                                        const cupdlp_float * __restrict__ x,
                                        const cupdlp_float * __restrict__ cost,
                                        const cupdlp_float * __restrict__ ATy,
                                        const cupdlp_float * __restrict__ lb,
                                        const cupdlp_float * __restrict__ ub,
                                        cupdlp_float dPrimalStep, int nCols) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nCols; i += gridDim.x * blockDim.x) {
    xUpdate[i] = min(max(cupdlp_fma_rn(dPrimalStep, ATy[i] - cost[i], x[i]), lb[i]), ub[i]);
  }
}

// yUpdate = proj(y + dDualStep * (b - 2 AxUpdate + Ax))
__global__ void dual_grad_step_kernel(cupdlp_float * __restrict__ yUpdate,
                                      const cupdlp_float * __restrict__ y,
                                      const cupdlp_float * __restrict__ b,
                                      const cupdlp_float * __restrict__ Ax,
                                      const cupdlp_float * __restrict__ AxUpdate,
                                      cupdlp_float dDualStep, int nRows, int nEqs) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nRows; i += gridDim.x * blockDim.x) {
    cupdlp_float upd = cupdlp_fma_rn(dDualStep, b[i] - 2 * AxUpdate[i] + Ax[i], y[i]);
    yUpdate[i] = i >= nEqs ? max(upd, 0.0) : upd;
  }
}

/*
// z = x - y
__global__ void naive_sub_kernel(cupdlp_float *z, const cupdlp_float *x,
                                 const cupdlp_float *y, int n) {
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    z[i] = x[i] - y[i];
  }
}
*/


#define QUARTER_WARP_REDUCE_2(val1, val2) { \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 4); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 4); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 2); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 2); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 1); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 1); \
}

#define FULL_WARP_REDUCE_2(val1, val2) { \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 16); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 16); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 8); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 8); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 4); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 4); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 2); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 2); \
  val1 += __shfl_down_sync(0xFFFFFFFF, val1, 1); \
  val2 += __shfl_down_sync(0xFFFFFFFF, val2, 1); \
}

// assumes block size = 256, warp size = 32
__global__ void movement_1_kernel(cupdlp_float * __restrict__ res_x, cupdlp_float * __restrict__ res_y,
                                  const cupdlp_float * __restrict__ xUpdate, const cupdlp_float * __restrict__ x,
                                  const cupdlp_float * __restrict__ atyUpdate, const cupdlp_float * __restrict__ aty,
                                  int nCols) {

  __shared__ cupdlp_float shared_x[32];
  __shared__ cupdlp_float shared_y[32];
  cupdlp_float val_x = 0.0;
  cupdlp_float val_y = 0.0;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nCols; i += blockDim.x * gridDim.x) {
      cupdlp_float dx = xUpdate[i] - x[i];
      cupdlp_float day = atyUpdate[i] - aty[i];
      val_x = cupdlp_fma_rn(dx, dx, val_x);
      val_y = cupdlp_fma_rn(day, dx, val_y);
  }

  int lane = threadIdx.x % 32;
  int wid = threadIdx.x / 32;

  FULL_WARP_REDUCE_2(val_x, val_y)
  if (lane == 0) {
    shared_x[wid] = val_x;
    shared_y[wid] = val_y;
  }
  __syncthreads();

  if (wid == 0) {
    val_x = (threadIdx.x < blockDim.x / 32) ? shared_x[lane] : 0.0;
    val_y = (threadIdx.x < blockDim.x / 32) ? shared_y[lane] : 0.0;
    QUARTER_WARP_REDUCE_2(val_x, val_y)
    if (threadIdx.x == 0) {
      res_x[blockIdx.x] = val_x;
      res_y[blockIdx.x] = val_y;
    }
  }
}

#define QUARTER_WARP_REDUCE(val) { \
  val += __shfl_down_sync(0xFFFFFFFF, val, 4); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 2); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 1); \
}

#define HALF_WARP_REDUCE(val) { \
  val += __shfl_down_sync(0xFFFFFFFF, val, 8); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 4); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 2); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 1); \
}

#define FULL_WARP_REDUCE(val) { \
  val += __shfl_down_sync(0xFFFFFFFF, val, 16); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 8); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 4); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 2); \
  val += __shfl_down_sync(0xFFFFFFFF, val, 1); \
}

// assumes: block size = 256, warp size = 32
__global__ void movement_2_kernel(cupdlp_float * __restrict__ res,
                                  const cupdlp_float * __restrict__ yUpdate, const cupdlp_float * __restrict__ y,
                                  int nRows) {

  __shared__ cupdlp_float shared[32];
  cupdlp_float val = 0.0;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nRows; i += blockDim.x * gridDim.x) {
      cupdlp_float d = yUpdate[i] - y[i];
      val = cupdlp_fma_rn(d, d, val);
  }

  int lane = threadIdx.x % 32;
  int wid = threadIdx.x / 32;

  FULL_WARP_REDUCE(val)
  if (lane == 0) {
    shared[wid] = val;
  }
  __syncthreads();

  if (wid == 0) {
    val = (threadIdx.x < blockDim.x / 32) ? shared[lane] : 0.0;
    QUARTER_WARP_REDUCE(val)
    if (threadIdx.x == 0) {
      res[blockIdx.x] = val;
    }
  }
}

// assumes: block size = 512, warp size = 32
__global__ void sum_kernel(cupdlp_float * __restrict__ res, const cupdlp_float * __restrict__ x, int n) {

  __shared__ cupdlp_float shared[32];
  cupdlp_float val = 0.0;
  for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
      val += x[i];
  }

  int lane = threadIdx.x % 32;
  int wid = threadIdx.x / 32;

  FULL_WARP_REDUCE(val)
  if (lane == 0) {
    shared[wid] = val;
  }
  __syncthreads();

  if (wid == 0) {
    val = (threadIdx.x < blockDim.x / 32) ? shared[lane] : 0.0;
    HALF_WARP_REDUCE(val)
    if (threadIdx.x == 0) {
      res[blockIdx.x] = val;
    }
  }
}
